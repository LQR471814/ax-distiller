#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cuco/static_map.cuh>
#include <hip/hip_runtime.h>
using namespace std;

//custom value
typedef struct Node {
    uint64_t fullkey;    //the key added up from all previous ancestors
    uint64_t hash;       //the current hash of this node and its subtree
    uint64_t firstChild;
    uint64_t nextSibling;
} Node;

// Helper function to check CUDA errors


int main() {
    auto myFile = fopen("buffer.bin", "rb");
    if (myFile == nullptr) {
        cerr << "Failed to open file buffer.bin" << endl;
        return 1;
    }
    
    uint64_t headerBuf[2];
    auto n = fread(headerBuf, sizeof(uint64_t), 2, myFile);
    if (n != 2) {
        cerr << "Failed to read file header" << endl;
        fclose(myFile);
        return 1;
    }
    
    uint64_t length = headerBuf[0];
    uint64_t rootHash = headerBuf[1];
    
    cout << "File header:" << endl;
    cout << "  Length: " << length << " nodes" << endl;
    cout << "  Root hash: 0x" << hex << rootHash << dec << endl << endl;
    
    // Allocate buffer for nodes
    Node* buffer = new Node[length];  // Use dynamic allocation for large arrays
    
    n = fread(buffer, sizeof(Node), length, myFile);
    if (n != length) {
        cerr << "Failed to read file contents. Expected " << length << " nodes, got " << n << endl;
        delete[] buffer;
        fclose(myFile);
        return 1;
    }
    
    fclose(myFile);
    cout << "Successfully read " << length << " nodes from file" << endl;
    
    // Copy to GPU
    Node* devicePtr;
    CUDA_CHECK(hipMalloc(&devicePtr, length * sizeof(Node)));
    CUDA_CHECK(hipMemcpy(devicePtr, buffer, length * sizeof(Node), hipMemcpyHostToDevice));
    
    cout << "Data copied to GPU" << endl;
    
    // *** COPY BACK FROM GPU TO CPU ***
    Node* resultBuffer = new Node[length];
    CUDA_CHECK(hipMemcpy(resultBuffer, devicePtr, length * sizeof(Node), hipMemcpyDeviceToHost));
    
    cout << "Data copied back from GPU" << endl << endl;
    
    // *** PRINT FILE CONTENTS ***
    cout << "Node contents (showing first 10 nodes):" << endl;
    cout << "Index | FullKey            | Hash               | FirstChild | NextSibling" << endl;
    cout << "------|--------------------|--------------------|------------|------------" << endl;
    
    size_t maxPrint = min(static_cast<size_t>(10), static_cast<size_t>(length));
    for (size_t i = 0; i < maxPrint; ++i) {
        printf("%5zu | 0x%016lx | 0x%016lx | %10lu | %11lu\n", 
               i, 
               resultBuffer[i].fullkey, 
               resultBuffer[i].hash, 
               resultBuffer[i].firstChild, 
               resultBuffer[i].nextSibling);
    }
    
    if (length > 10) {
        cout << "... (showing only first 10 of " << length << " total nodes)" << endl;
    }
    
    // Print some statistics
    cout << endl << "Statistics:" << endl;
    uint64_t nonZeroFullkeys = 0;
    uint64_t nonZeroHashes = 0;
    uint64_t nodesWithChildren = 0;
    uint64_t nodesWithSiblings = 0;
    
    for (size_t i = 0; i < length; ++i) {
        if (resultBuffer[i].fullkey != 0) nonZeroFullkeys++;
        if (resultBuffer[i].hash != 0) nonZeroHashes++;
        if (resultBuffer[i].firstChild != 0) nodesWithChildren++;
        if (resultBuffer[i].nextSibling != 0) nodesWithSiblings++;
    }
    
    cout << "  Non-zero fullkeys: " << nonZeroFullkeys << " / " << length << endl;
    cout << "  Non-zero hashes: " << nonZeroHashes << " / " << length << endl;
    cout << "  Nodes with children: " << nodesWithChildren << " / " << length << endl;
    cout << "  Nodes with siblings: " << nodesWithSiblings << " / " << length << endl;
    
    // Verify data integrity (compare original vs copied-back data)
    bool dataMatches = true;
    for (size_t i = 0; i < length; ++i) {
        if (buffer[i].fullkey != resultBuffer[i].fullkey ||
            buffer[i].hash != resultBuffer[i].hash ||
            buffer[i].firstChild != resultBuffer[i].firstChild ||
            buffer[i].nextSibling != resultBuffer[i].nextSibling) {
            dataMatches = false;
            break;
        }
    }
    
    cout << endl << "Data integrity check: " << (dataMatches ? "PASSED" : "FAILED") << endl;
    
    // Cleanup
    delete[] buffer;
    delete[] resultBuffer;
    CUDA_CHECK(hipFree(devicePtr));
    
    return 0;
}