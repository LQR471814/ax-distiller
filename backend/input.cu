// #include <iostream>
// // #include <fstream>
// #include <string>
// #include <cuco/static_map.cuh>
// #include <hip/hip_runtime.h>
// using namespace std;
#include <lib/hashmap.hpp>
#include <iostream>

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    } while(0)
//custom value
int main() {
  //TO DO add a file system queue to proccess each file individually
  auto myFile = fopen("buffer.bin", "rb");
    if (myFile == nullptr) {
        cerr << "Failed to open file buffer.bin" << endl;
        return 1;
    }
    
    uint64_t headerBuf[2];
    auto n = fread(headerBuf, sizeof(uint64_t), 2, myFile);
    if (n != 2) {
        cerr << "Failed to read file header" << endl;
        fclose(myFile);
        return 1;
    }
    
    uint64_t length = headerBuf[0];
    uint64_t rootHash = headerBuf[1];
    
    cout << "File header:" << endl;
    cout << "  Length: " << length << " nodes" << endl;
    cout << "  Root hash: 0x" << hex << rootHash << dec << endl << endl;

    Node* buffer = new Node[length];  // Use dynamic allocation for large arrays
  return 0;
}